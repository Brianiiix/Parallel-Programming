#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    float x = lowerX + i * stepX;
    float y = lowerY + j * stepY;

    int index = j * resX + i;
    d_img[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int* d_img;
    size_t size = resX * resY * sizeof(int);
    hipMalloc(&d_img, size);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);
    hipDeviceSynchronize();

    int* h_img = (int*)malloc(size);
    hipMemcpy(h_img, d_img, size, hipMemcpyDeviceToHost);
	memcpy(img, h_img, size);

	hipFree(d_img);
    free(h_img);
}
